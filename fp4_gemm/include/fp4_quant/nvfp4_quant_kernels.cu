#include <torch/all.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_fp8.h>

#include "cuda_utils.h"

#define ELTS_PER_THREAD 8

constexpr int CVT_FP4_ELTS_PER_THREAD = 8;
constexpr int CVT_FP4_SF_VEC_SIZE = 16;