#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_fp8.h>
#include <hip/hip_fp4.h>

__inline__ __device__ __hip_fp4_e2m1 cvt_bf16_to_nv_fp4_e2m1(const __hip_bfloat16 in) {
  __hip_fp4_e2m1 tmp = __hip_fp4_e2m1(in);
  return tmp;
}

__global__  void test_fun(__hip_bfloat16* a, __hip_fp4_e2m1* b, int n) {
  int tid = threadIdx.x;
  for (int i = tid; i < n; i += blockIdx.x) {
    b[i] = cvt_bf16_to_nv_fp4_e2m1(a[i]);
  }
}

int main() {
  int n = 128;
  __hip_bfloat16* a;
  size_t size_a = n * sizeof(__hip_bfloat16);
  for (int i = 0; i < 2; i++) {
      a[i] = i;
  }
  a = (__hip_bfloat16*)malloc(size_a);

  __hip_fp4_e2m1* b;
  size_t size_b = n * sizeof(__hip_fp4_e2m1);
  b = (__hip_fp4_e2m1*)malloc(size_b);

  __hip_bfloat16* da;
  __hip_fp4_e2m1* db;
  hipError_t err = hipMalloc(&da, size_a);
  err = hipMalloc(&db, size_b);
  hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
    return -1;
  }

  dim3 grid(1);
  dim3 block(128);

  test_fun<<<grid, block>>>(da, db, n);

  hipMemcpy(b, db, size_b, hipMemcpyDeviceToHost);

  for (int i = 0; i < 2; i++) {
    printf("%d \n", b[i]);
  }

  // 使用完毕后释放内存
  hipFree(da);
  return 0;
}